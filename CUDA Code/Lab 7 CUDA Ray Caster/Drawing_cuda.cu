#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_


#include <helper_gl.h>
#include <GL/freeglut.h>

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_vector_types.h>

 // includes, cuda
#include <hip/hip_runtime_api.h>

//header files for raycasting from Ray Tracing In One Weekend
#include <hitable.h>
#include <hitable_list.h>
#include <ray.h>
#include <sphere.h>
#include <vec3.h>

typedef unsigned int uint;
typedef unsigned char uchar;


hipArray* d_imageArray = 0;

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}
__device__ vec3 castRay(const ray& r, hitable** world) {
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5f * vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
    }
    else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f * (unit_direction.y() + 1.0f);
        return (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
    }
}
__global__ void create_world(hitable** d_list, hitable** d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int val = 12;
        float x = -4;
        float y = 0;
        //*(d_list) = new sphere(vec3(0, 0, -1), 0.5);
    
        for (int i = 0; i < val - 1; i++)
        {
            
            *(d_list + i) = new sphere(vec3(x, y, -1), 0.5);
            
            x = x + 0.5;
         

            if (i % 2 == 0)
            {
                x = x + 1;
                //x = x - 1;
            }
            else
            {
                //x = x / 2;
                //x = x + 1;
                y = y + 0.5;
            }
            
            
        }
        *(d_list + 11) = new sphere(vec3(0, -100.5, -1), 100);
        *d_world = new hitable_list(d_list, val);
    }
}
__global__ void free_world(hitable** d_list, hitable** d_world) {
    delete* (d_list);
    delete* (d_list + 1);
    delete* d_world;
}


__global__ void d_render(uchar4* d_output, uint width, uint height, hitable **d_world) {
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint i = __umul24(y, width) + x;

    float u = x / (float)width; //---> [0,1] x [0,1]
    float v = y / (float)height;

    u = 2.0f * u - 1.0f; //---> [-1,1] x [-1, 1]
    v = -(2.0f * v - 1.0f);

    u *= width / (float)height;

    u *= 2.0;
    v *= 2.0;

    vec3 eye = vec3(0, 0.5, 1.5);
    float distFrEye2Img = 1.0;;
    if ((x < width) && (y < height))
    {
        //for each pixel
        vec3 pixelPos = vec3(u, v, eye.z() - distFrEye2Img);

        //fire a ray
        ray r;
        r.O = eye;

        r.DIR = pixelPos - eye; //view direction along the negative axis
        
        vec3 col = castRay(r, d_world);
        float red = col.x();
        float green = col.y();
        float blue = col.z();
        d_output[i] = make_uchar4(red * 255, green * 255, blue * 255, 0);
    }

    

    //if ((x < width) && (y < height)) {
    //    /*int cx = width / 2;
    //    int cy = height / 2;
    //    float radius = min(width, height) / 4;*/

    //    float cx = 0.0f;
    //    float cy = 0.0f;
    //    float radius = 0.25f;

    //    float u = x / (float)width;
    //    float v = y / (float)height;

    //    u = 2.0f * u - 1.0f;
    //    v = -(2.0f * v - 1.0f);

    //    u *= width / (float)height;



    //    //float2 z = { u, v };
    //    //float2 T = { 0.1, 0.7 };
    //    //float r = 0;
    //    //float c = 1.0f;
    //    //for (int i = 0; i < 30; i++)
    //    //{
    //    //    //z = { z.x * z.x - z.y * z.y, 2.0f * z.x * z.y };
    //    //    //z += T;

    //    //    float tempX = z.x * z.x - z.y * z.y;
    //    //    float tempY = 2.0f * z.x * z.y;

    //    //    z.x = tempX + T.x;
    //    //    z.y = tempY + T.y;

    //    //    r = sqrt(z.x * z.x + z.y * z.y);
    //    //    if (r > 5.0)
    //    //    {
    //    //        c = 0.0;
    //    //        break;
    //    //    }
    //    //}


    //    //int distanceSquare = ((cx - u) * (cx - u)) + ((cy - v) * (cy - v));
    //    float distanceSquare = (u - cx) * (u - cx) + (v - cy) * (v - cy);

    //    //int c = ((((x & 0x1a) == 0) ^ ((y & 0x1a)) == 0)) * 255;
    //    //if (c == 255)
    //    if (distanceSquare <= radius * radius)
    //    //if (c == 0.0f)
    //    {
    //        d_output[i] = make_uchar4(0, 0, 0xff, 0xff);
    //    }
    //    else
    //    {
    //        d_output[i] = make_uchar4(0, 0, 0, 0);
    //    }

    //}
}


extern "C" void freeTexture() {

    checkCudaErrors(hipFreeArray(d_imageArray));
}

// render image using CUDA
extern "C" void render(int width, int height, dim3 blockSize, dim3 gridSize,
    uchar4* output) {

    // make our world of hitables
    hitable** d_list;
    checkCudaErrors(hipMalloc((void**)&d_list, 2 * sizeof(hitable*)));
    hitable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
    create_world << <1, 12 >> > (d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    d_render << <gridSize, blockSize >> > (output, width, height, d_world);
    getLastCudaError("kernel failed");

    /*d_render << <gridSize, blockSize >> > (output, width, height);


    getLastCudaError("kernel failed");*/
}


#endif